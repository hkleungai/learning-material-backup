#include <stdio.h>
#include <hip/hip_runtime.h>

#define M_SIZE	   32
#define BLOCK_SIZE 16

//Calculate the multiplication of two 32*32 matrices A and B in gpu and store the result in C.
//Each block calculate 16*16 submaxtrix of C.
__global__ void Mul(int *A,int *B,int *C,int width_A,int width_B)
{
        // Block index
    	int bx = blockIdx.x;
        int by = blockIdx.y;

        // Thread index
        int tx = threadIdx.x;
        int ty = threadIdx.y;

        //Index of the first sub-matrix of A and B processed by the block
        int aBegin=width_A*BLOCK_SIZE*by;
        int bBegin=BLOCK_SIZE*bx;

        //Csub of each thread stores corresponding element of the block sub-matrix.
        int Csub=0;
        int a=aBegin,b=bBegin;

	// Shared memory for the sub-matrix of A
	__shared__ int As[BLOCK_SIZE][BLOCK_SIZE];
	// Shared memory for the sub-matrix of B
	__shared__ int Bs[BLOCK_SIZE][BLOCK_SIZE];

	for (;a<aBegin+width_A-1;a+=BLOCK_SIZE,b+=BLOCK_SIZE*width_B) 
	{
        	// Load the matrices from global memory to shared memory;
        	// each thread loads one element of each matrix
        	As[ty][tx] = A[a+width_A*ty + tx];
        	Bs[ty][tx] = B[b+width_B*ty + tx];

       		// Synchronize to make sure the matrices are loaded
         	__syncthreads();

        	for (int k=0;k<BLOCK_SIZE;k++)
        	Csub+=As[ty][k]*Bs[k][tx];

        	//Make sure computation is done before loading new matrices for global memory.
        	__syncthreads();
	}

	//Write the block sub-matrix to global memory.
	int c=width_B*BLOCK_SIZE*by+BLOCK_SIZE*bx;
	C[c+width_B*ty+tx] = Csub;
}

int main()
{
	int size = M_SIZE*M_SIZE*sizeof(int);
	int *h_A=(int*)malloc(size);
	int *h_B=(int*)malloc(size);
	int *h_C=(int*)malloc(size);

	int *d_A,*d_B,*d_C;
	int i, j;
	//Intialize A,B,C
	for(i=0;i<M_SIZE*M_SIZE;i++)
	{
        	h_A[i]=1;
        	h_B[i]=1;
        	h_C[i]=0;
	}

	//Allocate the memory in GPU to store the content of A,B,C
	hipMalloc((void **)&d_A, size);
	hipMalloc((void **)&d_B, size);
	hipMalloc((void **)&d_C, size);

	//Copy h_A, h_B to d_A,d_B
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

	//Allocate 4 blocks and 256 threads per block.
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(2,2);
	Mul<<<dimGrid,dimBlock>>>(d_A, d_B, d_C, M_SIZE, M_SIZE);

	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

	//print the result
	for(i=0; i<M_SIZE; i++)
	{
		for(j=0; j<M_SIZE; j++)
		{
			printf("%d ", h_C[i*M_SIZE+j]);
		}
		printf("\n");
	}
	printf("Press Enter to stop\n");
	getchar();

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	free(h_A);
	free(h_B);
	free(h_C);

	return 0;
}

