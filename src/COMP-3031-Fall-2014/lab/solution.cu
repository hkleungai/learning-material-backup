#include <stdio.h>
#include <hip/hip_runtime.h>

#define MATRIX_SIZE 64

__global__ void Square(int *A)
{
	// Block index
	/************Add your code***********/
	int bx = blockIdx.x; 
        int by = blockIdx.y; 
	// Thread index
	/************Add your code***********/
	int tx = threadIdx.x; 
        int ty = threadIdx.y;
	//Calculation
	/************Add your code***********/
	int row = 32*by+ty;
	int column = 32*bx+tx;
	int index = MATRIX_SIZE*row+column;

	A[index]=A[index]*A[index];
}

int main()
{
	int size = MATRIX_SIZE*MATRIX_SIZE*sizeof(int);
	int *h_A = (int *)malloc(size);
	int *d_A;
	int i;

	//Intialize A
	for(i=0;i<MATRIX_SIZE*MATRIX_SIZE;i++)
	{
	                h_A[i] = 2;
	}

	//Allocate the memory in GPU to store the content of A
	/************Add your code***********/
	hipMalloc((void **)&d_A,size);

	//Copy h_A to d_A
	/************Add your code***********/
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

	//Allocate blocks and 32*32 threads per block.
	/************Add your code***********/
	dim3 dimBlock(32, 32);
        dim3 dimGrid(2,2);	


	//Run the kernel
	/************Add your code***********/
	Square<<<dimGrid,dimBlock>>>(d_A);

	//Copy the result to CPU
	/************Add your code***********/
	hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);

	//free GPU memory for d_A
	/************Add your code***********/
	hipFree(d_A);

	//free Host Memory
	free(h_A);

	return 0;
}
