#include "hip/hip_runtime.h"
/**
 *  Demo code of Cuda programming lecture
 *  
 *  This programme is a simple implementation of vector addition in CUDA 
 *   Windows Version
 *
 *      hupmscy@HKUST, Nov. 28, 2012
 *      Revised by xjia@HKUST, Feb. 7, 2015
 *
 */


#include <cstdlib>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <>

#ifdef WIN32
#include <ctime>
#include <Windows.h>
int gettimeofday(struct timeval *tp, void *tzp)
{
    time_t clock;
    struct tm tm;
    SYSTEMTIME wtm;

    GetLocalTime(&wtm);
    tm.tm_year = wtm.wYear - 1900;
    tm.tm_mon = wtm.wMonth - 1;
    tm.tm_mday = wtm.wDay;
    tm.tm_hour = wtm.wHour;
    tm.tm_min = wtm.wMinute;
    tm.tm_sec = wtm.wSecond;
    tm.tm_isdst = -1;

    clock = mktime(&tm);
    tp->tv_sec = clock;
    tp->tv_usec = wtm.wMilliseconds * 1000;
    return (0);

}
#endif
// Device code
__global__ void VecAdd(int* A, int* B, int* C)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    C[i] = A[i] + B[i];
}

// Host code
int main()
{
    int *h_A, *h_B, *h_C, *d_A, *d_B, *d_C; 
    int N = 33554432;
    size_t size = N * sizeof(int);
   
    int threadsPerBlock = 1024;
    int blocksPerGrid = N / threadsPerBlock;

    //Time measurement
    timeval kernel_start, kernel_end;
    timeval global_start, global_end;
    float kernel_elapsed_time, global_elapsed_time; 
        
    // Allocate host memory
    h_A = (int*)malloc(size);
    h_B = (int*)malloc(size);
    h_C = (int*)malloc(size);
    
    //Initialization
    for (int i = 0; i < N; i++)
    {
        h_A[i] = i;
        h_B[i] = i;
    }
    
    // Allocate device memory
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);
    
    //Start global timer
    gettimeofday(&global_start, NULL);  

    // Copy vectors from host memory to device memory
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    
    //Start kernel timer
    gettimeofday(&kernel_start, NULL);      
    
    // Invoke kernel
    VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C);
    
    //Since kernel launch is asynchronized, block the host code until the kernel finishes
    hipDeviceSynchronize();
    
    //End kernel timer  
    gettimeofday(&kernel_end, NULL);
    
        
    // Copy result from device memory to host memory
    // h_C contains the result in host memory
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);      
   
    //hipMemcpy is synchronized, no barrier is needed here
    //Stop global timer
    gettimeofday(&global_end, NULL);                        

    //get kernel elapsed time
    kernel_elapsed_time = 1000*(kernel_end.tv_sec - kernel_start.tv_sec) + (float)(kernel_end.tv_usec - kernel_start.tv_usec)/1000;
    //get global elapsed time
    global_elapsed_time = 1000*(global_end.tv_sec - global_start.tv_sec) + (float)(global_end.tv_usec - global_start.tv_usec)/1000;
    
    printf("elapsed time of gpu vector addition(time cost by data transfer between host and device is excluded): %.2f ms\n", kernel_elapsed_time);
    printf("elapsed time of gpu vector addition(time cost by data transfer between host and device is included): %.2f ms\n", global_elapsed_time);

   //Free host memory
   free(h_A);
   free(h_B);
   free(h_C);
   
   //Free device memory
   hipFree(d_A);
   hipFree(d_B);
   hipFree(d_C);

   getchar();
}

