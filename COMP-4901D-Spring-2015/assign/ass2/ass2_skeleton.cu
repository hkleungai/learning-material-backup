#include "hip/hip_runtime.h"
/* skeleton code for assignment2 COMP4901D
   xjia@ust.hk 2015/03
 */
#include <iostream>
#include <cstdio>
#include <cmath>
#include <hip/hip_runtime.h>
#include <>
#include <thrust/sort.h>
#include <thrust/device_vector.h>
using namespace std;

const int TILE_WIDTH = 1024;
__global__ void mergeJoin(int *key1, float *value1, int *key2, float *value2,int N1,int N2,int *result)
{
  //chunk of keys in the shared memory
  __shared__ int s_key[TILE_WIDTH];

  //start and end positions of the elements in the matching chunk
  __shared__ int start;
  __shared__ int end;
  
  //add you code here
}

int main()
{
  freopen("in.txt","r",stdin);
  int *h_key1, *h_key2, *d_key1, *d_key2;
  float *h_value1, *h_value2, *d_value1, *d_value2;
  int *h_result, *d_result;
  int N1,N2;

  scanf("%d%d",&N1,&N2);

  h_key1 = (int*)malloc(N1 * sizeof(int));
  h_key2 = (int*)malloc(N2 * sizeof(int));
  h_value1 = (float*)malloc(N1 * sizeof(float));
  h_value2 = (float*)malloc(N2 * sizeof(float));
  h_result = (int*)malloc(N1 * sizeof(int));

  hipMalloc(&d_key1, N1 * sizeof(int));
  hipMalloc(&d_key2, N2 * sizeof(int));
  hipMalloc(&d_value1, N1 * sizeof(float));
  hipMalloc(&d_value2, N2 * sizeof(float));
  hipMalloc(&d_result, N1 * sizeof(int));

  for(int i = 0; i < N1; ++i)
	scanf("%d%f",&h_key1[i],&h_value1[i]);

  for(int i = 0; i < N2; ++i)
	scanf("%d%f",&h_key2[i],&h_value2[i]);


  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  
  memset(h_result,-1,sizeof(int) * N1);
  hipMemcpy(d_key1,h_key1, sizeof(int) * N1, hipMemcpyHostToDevice);
  hipMemcpy(d_result,h_result, sizeof(int) * N1, hipMemcpyHostToDevice);
  hipMemcpy(d_key2,h_key2, sizeof(int) * N2, hipMemcpyHostToDevice);
  hipMemcpy(d_value1,h_value1, sizeof(float) * N1, hipMemcpyHostToDevice);
  hipMemcpy(d_value2,h_value2, sizeof(float) * N2, hipMemcpyHostToDevice);

  hipEventRecord(start,0);

  thrust::device_ptr<int> dev_key1(d_key1);
  thrust::device_ptr<int> dev_key2(d_key2);
  thrust::device_ptr<float> dev_value1(d_value1);
  thrust::device_ptr<float> dev_value2(d_value2);
  thrust::sort_by_key(dev_key1,dev_key1 + N1, dev_value1);
  thrust::sort_by_key(dev_key2,dev_key2 + N2, dev_value2);
  
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  float ElapsedTime;
  hipEventElapsedTime(&ElapsedTime,start,stop);
  printf("Sort Elapsed Time: %.3f ms\n",ElapsedTime);

  dim3 grid(ceil(double(N2)/1024));
  dim3 block(1024);


  hipEventRecord(start,0);
  mergeJoin<<<grid,block>>>(d_key1,d_value1,d_key2,d_value2,N1,N2,d_result);
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&ElapsedTime,start,stop);
  printf("kernel mergeJoin Elapsed Time: %.3f ms\n",ElapsedTime);

  hipMemcpy(h_result,d_result,sizeof(int) * N1, hipMemcpyDeviceToHost);
  hipMemcpy(h_key1,d_key1,sizeof(int) * N1, hipMemcpyDeviceToHost);
  hipMemcpy(h_value1,d_value1,sizeof(float) * N1, hipMemcpyDeviceToHost);
  hipMemcpy(h_value2,d_value2,sizeof(float) * N2, hipMemcpyDeviceToHost);
  
  int matched = 0;
  freopen("out.txt","w",stdout);
  for(int i = 0;i < N1; ++i)
  {
	if(h_result[i] == -1)
	  continue;
	matched++;
	printf("Key %d\nValue1 %.2f Value2 %.2f\n\n",h_key1[i],h_value1[i],h_value2[h_result[i]]);
  }
  printf("Matched %d\n",matched);
  fclose(stdout);

  free(h_key1);
  free(h_key2);
  free(h_value1);
  free(h_value2);
  free(h_result);

  hipFree(d_key1);
  hipFree(d_key2);
  hipFree(d_value1);
  hipFree(d_value2);
  hipFree(d_result);
  return 0;
}

