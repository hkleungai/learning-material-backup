#include <iostream>
#include <cstdio>
#include <cmath>
#include <hip/hip_runtime.h>

using namespace std;

const int TILE_WIDTH = 1024;
__global__ void equiJoin(int *key1, float *value1, int *key2, float *value2,int N1,int N2,int *result)
{
  /* fill in your code here */
}
__global__ void equiJoinTiled(int *key1, float *value1, int *key2, float *value2,int N1,int N2,int *result)
{
  __shared__ int s_key[TILE_WIDTH];
  /* fill in your code here */
}

int main()
{
  freopen("in.txt","r",stdin);
  int *h_key1, *h_key2, *d_key1, *d_key2;
  float *h_value1, *h_value2, *d_value1, *d_value2;
  int *h_result1, *h_result2, *d_result1, *d_result2;
  int N1,N2;

  scanf("%d%d",&N1,&N2);
  h_key1 = (int*)malloc(N1 * sizeof(int));
  h_key2 = (int*)malloc(N2 * sizeof(int));
  h_value1 = (float*)malloc(N1 * sizeof(float));
  h_value2 = (float*)malloc(N2 * sizeof(float));
  
  h_result1 = (int*)malloc(N1 * sizeof(int));
  h_result2 = (int*)malloc(N1 * sizeof(int));

  hipMalloc(&d_key1, N1 * sizeof(int));
  hipMalloc(&d_key2, N2 * sizeof(int));
  hipMalloc(&d_value1, N1 * sizeof(float));
  hipMalloc(&d_value2, N2 * sizeof(float));
  hipMalloc(&d_result1, N1 * sizeof(int));
  hipMalloc(&d_result2, N1 * sizeof(int));

  for(int i = 0; i < N1; ++i)
	scanf("%d%f",&h_key1[i],&h_value1[i]);

  for(int i = 0; i < N2; ++i)
	scanf("%d%f",&h_key2[i],&h_value2[i]);

  hipMemcpy(d_key1,h_key1, sizeof(int) * N1, hipMemcpyHostToDevice);
  hipMemcpy(d_key2,h_key2, sizeof(int) * N2, hipMemcpyHostToDevice);
  hipMemcpy(d_value1,h_value1, sizeof(float) * N1, hipMemcpyHostToDevice);
  hipMemcpy(d_value2,h_value2, sizeof(float) * N2, hipMemcpyHostToDevice);

  dim3 grid1(ceil(double(N1/1024)));
  dim3 block1(1024);

  dim3 grid2(ceil(double(N1/TILE_WIDTH)));
  dim3 block2(TILE_WIDTH);
  
  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  //test kernel equiJoin
  hipEventRecord(start,0);
  
  equiJoin<<<grid1,block1>>>(d_key1,d_value1,d_key2,d_value2,N1,N2,d_result1);
  
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  float ElapsedTime;
  hipEventElapsedTime(&ElapsedTime,start,stop);
  printf("kernel equiJoin Elapsed Time: %.3lf ms\n",ElapsedTime);

  hipMemcpy(h_result1,d_result1,sizeof(int) * N1, hipMemcpyDeviceToHost);

  //test kernel equiJoinTiled
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start,0);
 
  equiJoinTiled<<<grid2,block2>>>(d_key1,d_value1,d_key2,d_value2,N1,N2,d_result2);

  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&ElapsedTime,start,stop);
  printf("kernel equiJoinTiled Elapsed Time: %.3lf ms\n",ElapsedTime);
  
  hipMemcpy(h_result2,d_result2,sizeof(int) * N1, hipMemcpyDeviceToHost);

  //check whether h_result1 is same as h_result2
  bool same = true;
  for(int i = 0; i < N1; ++i)
  {
	if(h_result1[i] != h_result2[i])
	{
	  same = false;
	  break;
	}
  }
  if(!same)
  {
	printf("Error!\n");
	return 0;
  }
  int matched = 0;
  freopen("out.txt","w",stdout);
  for(int i = 0;i < N1; ++i)
  {
	if(h_result1[i] == -1)
	  continue;
	matched++;
	printf("Key %d\nValue1 %.2f Value2 %.2f\n\n",h_key1[i],h_value1[i],h_value2[h_result1[i]]);
  }
  printf("Matched %d\n",matched);
  fclose(stdout);
  
  free(h_key1);
  free(h_key2);
  free(h_value1);
  free(h_value2);
  free(h_result1);
  free(h_result2);

  hipFree(h_key1);
  hipFree(h_key2);
  hipFree(h_value1);
  hipFree(h_value2);
  hipFree(h_result1);
  hipFree(h_result2);
  return 0;
}

