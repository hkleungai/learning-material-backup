#include "hip/hip_runtime.h"
/* 
   skeleton code for assignment3 COMP4901D
   Hash Join
   xjia@ust.hk 2015/04/15
 */
#include <iostream>
#include <cstdio>
#include <cmath>
#include <hip/hip_runtime.h>
#include <>
#include <thrust/sort.h>
#include <thrust/device_vector.h>
using namespace std;

const int numBits = 6;
const int totalBits = 19;
const int numPart = 1 << numBits;
const int numPerPart = 1 << (totalBits - numBits);
const int mask = (1 << numBits) - 1;
const int numThreads = 128;
const int numBlocks = 512;

/*
   return the partition ID of the input element
*/
  __device__
int getPartID(int element)
{
  element >>= (totalBits - numBits);
  return element & mask;
}

/*
	input: d_key[], array size N
	output: d_pixArray[]
	funciton: for input array d_key[] with size N, return the partition ID array d_pixArray[]
*/
  __global__
void mapPart(int d_key[],int d_pidArray[],int N)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int threadNumber = blockDim.x * gridDim.x;

  while(tid < N)
  {
	d_pidArray[tid] = getPartID(d_key[tid]);
	tid += threadNumber;
  }
}

/*
   input: d_pidArray[], array size N
   output: d_Hist[] 
   function: calculate the histogram d_Hist[] based on the partition ID array d_pidArray[]
*/
  __global__
void count_Hist(int d_Hist[],int d_pidArray[],int N)
{
  __shared__ int s_Hist[numThreads * numPart];
  int threadId = blockIdx.x * blockDim.x + threadIdx.x;
  int threadNumber = blockDim.x * gridDim.x;
  int offset = threadIdx.x * numPart;

  for(int i = 0; i < numPart; ++i)
	s_Hist[i + offset] = 0;

  for(int i = threadId; i < N; i += threadNumber)
	s_Hist[offset + d_pidArray[i]]++;

  for(int i = 0; i < numPart; ++i)
	d_Hist[i * threadNumber + threadId] = s_Hist[offset + i];
  __syncthreads();
}
/*
	input: d_pidArray[] (partition ID array), d_psSum[] (prefix sum of histogram), array size N
	output: d_loc[] (location array)
	function: for each element, calculate its corresponding location in the result array based on its partition ID and prefix sum of histogram
*/
  __global__
void write_Hist(int d_pidArray[],int d_psSum[],int d_loc[],int N)
{
  __shared__ int s_psSum[numThreads * numPart];
  int threadId = threadIdx.x + blockIdx.x * blockDim.x;
  int threadNumber = gridDim.x * blockDim.x;
  int offset = threadIdx.x * numPart;

  for(int i = 0; i < numPart; ++i)
	s_psSum[i + offset] = d_psSum[threadId + i * threadNumber];

  for(int i = threadId; i < N; i += threadNumber)
  {
	int pid = d_pidArray[i];
	d_loc[i] = s_psSum[pid + offset];
	s_psSum[pid + offset]++;
  }
}

/*
	input: d_psSum[] (prefix sum of histogram), array size N
	output: start position of each partition
	function: for each partition (chunck to be loaded in the join step), calculate its start position in the result array (the first element's position of this partition)
*/
  __global__
void getStartPos(int d_psSum[],int d_startPos[],int N)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int threadNumber = gridDim.x * blockDim.x;

  if(tid >= numPart)
	return;
  d_startPos[tid] = d_psSum[tid * threadNumber];
}

/*
    input: d_key[],d_value[],d_loc[],array size []
	output: out_key[],out_value[]
	function: rewrite the (key,value) pair to its corresponding position based on location array d_loc[]
*/
  __global__
void scatter(int d_key[],float d_value[],int out_key[],float out_value[],int d_loc[],int N)
{
  int threadId = threadIdx.x + blockIdx.x * blockDim.x;
  int threadNumber = blockDim.x * gridDim.x;

  while(threadId < N)
  {
	out_key[d_loc[threadId]] = d_key[threadId];
	out_value[d_loc[threadId]] = d_value[threadId];
	threadId += threadNumber;
  }
}

/*
	function: split the (key,value) array with size N, record the start position of each partition at the same time
*/
void split(int *d_key,float *d_value,int *d_startPos,int N)
{
  /* add your code here */
}

/*
	function: perform hash join on two (key,value) arrays 
*/   
  __global__
void join(int d_key1[],float d_value1[],int d_key2[],float d_value2[],int d_startPos1[],int d_startPos2[],int d_result[],int N1,int N2)
{
  /* add your code here */
}

void hashJoin(int *d_key1,float *d_value1,int *d_key2,float *d_value2,int N1,int N2,int *d_result)
{
  int *d_startPos1,*d_startPos2;
  hipMalloc(&d_startPos1,sizeof(int) * numPart);
  hipMalloc(&d_startPos2,sizeof(int) * numPart);
  split(d_key1,d_value1,d_startPos1,N1);
  split(d_key2,d_value2,d_startPos2,N2);

  dim3 grid(numPart);
  dim3 block(1024);

  join<<<grid,block>>>(d_key1,d_value1,d_key2,d_value2,d_startPos1,d_startPos2,d_result,N1,N2);
}
int main()
{
  freopen("in.txt","r",stdin);
  int *h_key1, *h_key2, *d_key1, *d_key2;
  float *h_value1, *h_value2, *d_value1, *d_value2;
  int *h_result, *d_result;
  int N1,N2;

  scanf("%d%d",&N1,&N2);

  h_key1 = (int*)malloc(N1 * sizeof(int));
  h_key2 = (int*)malloc(N2 * sizeof(int));
  h_value1 = (float*)malloc(N1 * sizeof(float));
  h_value2 = (float*)malloc(N2 * sizeof(float));
  h_result = (int*)malloc(N1 * sizeof(int));

  hipMalloc(&d_key1, N1 * sizeof(int));
  hipMalloc(&d_key2, N2 * sizeof(int));
  hipMalloc(&d_value1, N1 * sizeof(float));
  hipMalloc(&d_value2, N2 * sizeof(float));
  hipMalloc(&d_result, N1 * sizeof(int));

  for(int i = 0; i < N1; ++i)
	scanf("%d%f",&h_key1[i],&h_value1[i]);

  for(int i = 0; i < N2; ++i)
	scanf("%d%f",&h_key2[i],&h_value2[i]);

  memset(h_result,-1,sizeof(int) * N1);
  hipMemcpy(d_key1,h_key1, sizeof(int) * N1, hipMemcpyHostToDevice);
  hipMemcpy(d_result,h_result, sizeof(int) * N1, hipMemcpyHostToDevice);
  hipMemcpy(d_key2,h_key2, sizeof(int) * N2, hipMemcpyHostToDevice);
  hipMemcpy(d_value1,h_value1, sizeof(float) * N1, hipMemcpyHostToDevice);
  hipMemcpy(d_value2,h_value2, sizeof(float) * N2, hipMemcpyHostToDevice);
  
  hashJoin(d_key1,d_value1,d_key2,d_value2,N1,N2,d_result);

  hipMemcpy(h_result,d_result,sizeof(int) * N1, hipMemcpyDeviceToHost);
  hipMemcpy(h_key1,d_key1,sizeof(int) * N1, hipMemcpyDeviceToHost);
  hipMemcpy(h_key2,d_key2,sizeof(int) * N2, hipMemcpyDeviceToHost);
  hipMemcpy(h_value1,d_value1,sizeof(float) * N1, hipMemcpyDeviceToHost);
  hipMemcpy(h_value2,d_value2,sizeof(float) * N2, hipMemcpyDeviceToHost);

  int matched = 0;
  freopen("out.txt","w",stdout);
  for(int i = 0;i < N1; ++i)
  {
	if(h_result[i] == -1)
	  continue;
	matched++;
	printf("Key %d\nValue1 %.2f Value2 %.2f\n\n",h_key1[i],h_value1[i],h_value2[h_result[i]]);
  }
  printf("Matched %d\n",matched);
  fclose(stdout);
  freopen("/dev/tty","w",stdout);
  
  free(h_key1);
  free(h_key2);
  free(h_value1);
  free(h_value2);
  free(h_result);

  hipFree(d_key1);
  hipFree(d_key2);
  hipFree(d_value1);
  hipFree(d_value2);
  hipFree(d_result);

  hipDeviceReset();
  return 0;
}

