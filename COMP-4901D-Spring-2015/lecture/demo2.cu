

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__
void kernel1(int* d_data) {
	const int tid = blockDim.x*blockIdx.x + threadIdx.x;
	d_data[tid] += 1;
}


__global__
void kernel2(int* d_data, const int numElement) {
	const int tid = blockDim.x*blockIdx.x + threadIdx.x;
	const int nthread = blockDim.x*gridDim.x;
	const int numElementPerThread = numElement/nthread;
	const int start = tid*numElementPerThread;
	int end = start + numElementPerThread;
	
	for(int i = start; i < end; i++) {
		d_data[i] += 1;
	}
}

//
__global__
void kernel2_opt(int* d_data, const int numElement) {
	const int tid = blockDim.x*blockIdx.x + threadIdx.x;
	const int nthread = blockDim.x*gridDim.x;

	for(int i = tid; i < numElement; i += nthread) {
		d_data[i] += 1;
	}
}


void demo1() {

	const int numElement = 512*1024;	
	int* h_data = (int*)malloc(sizeof(int)*numElement);
	int* gold = (int*)malloc(sizeof(int)*numElement);
	for(int i = 0; i < numElement; i++) {
		h_data[i] = rand();
		gold[i] = h_data[i] + 1;
	}

	int* d_data;
	hipMalloc(&d_data, sizeof(int)*numElement);
	hipMemcpy(d_data, h_data, sizeof(int)*numElement, 
		   hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	kernel1<<<1024, 512>>>(d_data);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("Kernel elapsed time: %.3f ms\n", elapsedTime);


	printf("kernel1: %s\n", hipGetErrorString(hipGetLastError()));


	hipMemcpy(h_data, d_data, sizeof(int)*numElement, 
		   hipMemcpyDeviceToHost);
	hipFree(d_data);

	for(int i = 0; i < numElement; i++) {
		if(h_data[i] != gold[i]) {
			printf("!!!ERROR, TEST FAILED.\n");
			return;
		}
	}
	printf("Test pass...\n");

	free(h_data);
	free(gold);
	
}


void demo2(const int numElement) {

	printf("numElement = %d\n", numElement);

	int* h_data = (int*)malloc(sizeof(int)*numElement);
	int* gold = (int*)malloc(sizeof(int)*numElement);
	for(int i = 0; i < numElement; i++) {
		h_data[i] = rand();
		gold[i] = h_data[i];
	}



	int* d_data;
	hipMalloc(&d_data, sizeof(int)*numElement);
	hipMemcpy(d_data, h_data, sizeof(int)*numElement, 
		   hipMemcpyHostToDevice);
	float elapsedTime = 0.0f;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	/*CPU*/
	elapsedTime = 0.0f;
	hipEventRecord(start, 0);
	for(int i = 0; i < numElement; i++) {
		gold[i] += 1;
	}
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("CPU elapsed time: %.3f ms\n", elapsedTime);



	/*GPU method 1*/
	elapsedTime = 0.0f;
	hipEventRecord(start, 0);
	kernel2<<<1024, 512>>>(d_data, numElement);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("kernel2 elapsed time: %.3f ms\n", elapsedTime);
	printf("kernel2: %s\n", hipGetErrorString(hipGetLastError()));


	/*GPU method 2*/
	hipMemcpy(d_data, h_data, sizeof(int)*numElement, 
		   hipMemcpyHostToDevice);
	hipEventRecord(start, 0);
	kernel2_opt<<<1024, 512>>>(d_data, numElement);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("kernel2_opt elapsed time: %.3f ms\n", elapsedTime);
	printf("kernel2: %s\n", hipGetErrorString(hipGetLastError()));
	

	hipMemcpy(h_data, d_data, sizeof(int)*numElement, 
		   hipMemcpyDeviceToHost);
	hipFree(d_data);

	for(int i = 0; i < numElement; i++) {
		if(h_data[i] != gold[i]) {
			printf("!!!ERROR, TEST FAILED. i = %d: %d, %d\n", 
				i, h_data[i], gold[i]);
			return;
		}
	}
	printf("Test pass...\n");

	free(h_data);
	free(gold);
	
}




int main() {
	int numElement = 1*1024*1024;
	demo2(numElement); //execute once to warm up for performance measurement
	printf("\n\nstart ............................................\n");
	printf("demo2 started!\n");
	for(int i = numElement; i <= 32*1024*1024; i*=2) {
		demo2(i);
		printf("\n");
	}
	printf("demo1 started!\n");
	demo1();
	return EXIT_SUCCESS;
}

